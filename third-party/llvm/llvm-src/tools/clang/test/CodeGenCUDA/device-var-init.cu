#include "hip/hip_runtime.h"
// REQUIRES: nvptx-registered-target
// REQUIRES: amdgpu-registered-target

// Make sure we don't allow dynamic initialization for device
// variables, but accept empty constructors allowed by CUDA.

// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device -std=c++11 \
// RUN:     -fno-threadsafe-statics -emit-llvm -o - %s | FileCheck -check-prefixes=CHECK,NVPTX %s

// RUN: %clang_cc1 -triple amdgcn -fcuda-is-device -std=c++11 \
// RUN:     -fno-threadsafe-statics -emit-llvm -o - %s | FileCheck -check-prefixes=CHECK,AMDGCN %s

#ifdef __clang__
#include "Inputs/hip/hip_runtime.h"
#endif

// Use the types we share with Sema tests.
#include "Inputs/cuda-initializers.h"

__device__ int d_v;
// CHECK: @d_v = addrspace(1) externally_initialized global i32 0,
__shared__ int s_v;
// CHECK: @s_v = addrspace(3) global i32 undef,
__constant__ int c_v;
// CHECK: addrspace(4) externally_initialized global i32 0,

__device__ int d_v_i = 1;
// CHECK: @d_v_i = addrspace(1) externally_initialized global i32 1,

// trivial constructor -- allowed
__device__ T d_t;
// CHECK: @d_t = addrspace(1) externally_initialized global %struct.T zeroinitializer
__shared__ T s_t;
// CHECK: @s_t = addrspace(3) global %struct.T undef,
__constant__ T c_t;
// CHECK: @c_t = addrspace(4) externally_initialized global %struct.T zeroinitializer,

__device__ T d_t_i = {2};
// CHECK: @d_t_i = addrspace(1) externally_initialized global %struct.T { i32 2 },
__constant__ T c_t_i = {2};
// CHECK: @c_t_i = addrspace(4) externally_initialized global %struct.T { i32 2 },

// empty constructor
__device__ EC d_ec;
// CHECK: @d_ec = addrspace(1) externally_initialized global %struct.EC zeroinitializer,
__shared__ EC s_ec;
// CHECK: @s_ec = addrspace(3) global %struct.EC undef,
__constant__ EC c_ec;
// CHECK: @c_ec = addrspace(4) externally_initialized global %struct.EC zeroinitializer,

// empty destructor
__device__ ED d_ed;
// CHECK: @d_ed = addrspace(1) externally_initialized global %struct.ED zeroinitializer,
__shared__ ED s_ed;
// CHECK: @s_ed = addrspace(3) global %struct.ED undef,
__constant__ ED c_ed;
// CHECK: @c_ed = addrspace(4) externally_initialized global %struct.ED zeroinitializer,

__device__ ECD d_ecd;
// CHECK: @d_ecd = addrspace(1) externally_initialized global %struct.ECD zeroinitializer,
__shared__ ECD s_ecd;
// CHECK: @s_ecd = addrspace(3) global %struct.ECD undef,
__constant__ ECD c_ecd;
// CHECK: @c_ecd = addrspace(4) externally_initialized global %struct.ECD zeroinitializer,

// empty templated constructor -- allowed with no arguments
__device__ ETC d_etc;
// CHECK: @d_etc = addrspace(1) externally_initialized global %struct.ETC zeroinitializer,
__shared__ ETC s_etc;
// CHECK: @s_etc = addrspace(3) global %struct.ETC undef,
__constant__ ETC c_etc;
// CHECK: @c_etc = addrspace(4) externally_initialized global %struct.ETC zeroinitializer,

__device__ NCFS d_ncfs;
// CHECK: @d_ncfs = addrspace(1) externally_initialized global %struct.NCFS { i32 3 }
__constant__ NCFS c_ncfs;
// CHECK: @c_ncfs = addrspace(4) externally_initialized global %struct.NCFS { i32 3 }

// Regular base class -- allowed
__device__ T_B_T d_t_b_t;
// CHECK: @d_t_b_t = addrspace(1) externally_initialized global %struct.T_B_T zeroinitializer,
__shared__ T_B_T s_t_b_t;
// CHECK: @s_t_b_t = addrspace(3) global %struct.T_B_T undef,
__constant__ T_B_T c_t_b_t;
// CHECK: @c_t_b_t = addrspace(4) externally_initialized global %struct.T_B_T zeroinitializer,

// Incapsulated object of allowed class -- allowed
__device__ T_F_T d_t_f_t;
// CHECK: @d_t_f_t = addrspace(1) externally_initialized global %struct.T_F_T zeroinitializer,
__shared__ T_F_T s_t_f_t;
// CHECK: @s_t_f_t = addrspace(3) global %struct.T_F_T undef,
__constant__ T_F_T c_t_f_t;
// CHECK: @c_t_f_t = addrspace(4) externally_initialized global %struct.T_F_T zeroinitializer,

// array of allowed objects -- allowed
__device__ T_FA_T d_t_fa_t;
// CHECK: @d_t_fa_t = addrspace(1) externally_initialized global %struct.T_FA_T zeroinitializer,
__shared__ T_FA_T s_t_fa_t;
// CHECK: @s_t_fa_t = addrspace(3) global %struct.T_FA_T undef,
__constant__ T_FA_T c_t_fa_t;
// CHECK: @c_t_fa_t = addrspace(4) externally_initialized global %struct.T_FA_T zeroinitializer,


// Calling empty base class initializer is OK
__device__ EC_I_EC d_ec_i_ec;
// CHECK: @d_ec_i_ec = addrspace(1) externally_initialized global %struct.EC_I_EC zeroinitializer,
__shared__ EC_I_EC s_ec_i_ec;
// CHECK: @s_ec_i_ec = addrspace(3) global %struct.EC_I_EC undef,
__constant__ EC_I_EC c_ec_i_ec;
// CHECK: @c_ec_i_ec = addrspace(4) externally_initialized global %struct.EC_I_EC zeroinitializer,

// CHECK: @_ZZ2dfvE4s_ec = internal addrspace(3) global %struct.EC undef
// CHECK: @_ZZ2dfvE5s_etc = internal addrspace(3) global %struct.ETC undef

// CHECK: @_ZZ2dfvE11const_array = internal addrspace(4) constant [5 x i32] [i32 1, i32 2, i32 3, i32 4, i32 5]
// CHECK: @_ZZ2dfvE9const_int = internal addrspace(4) constant i32 123

// We should not emit global initializers for device-side variables.
// CHECK-NOT: @__cxx_global_var_init

// Make sure that initialization restrictions do not apply to local
// variables.
__device__ void df() {
  // NVPTX:  %[[ec:.*]] = alloca %struct.EC
  // NVPTX:  %[[ed:.*]] = alloca %struct.ED
  // NVPTX:  %[[ecd:.*]] = alloca %struct.ECD
  // NVPTX:  %[[etc:.*]] = alloca %struct.ETC
  // NVPTX:  %[[uc:.*]] = alloca %struct.UC
  // NVPTX:  %[[ud:.*]] = alloca %struct.UD
  // NVPTX:  %[[eci:.*]] = alloca %struct.ECI
  // NVPTX:  %[[nec:.*]] = alloca %struct.NEC
  // NVPTX:  %[[ned:.*]] = alloca %struct.NED
  // NVPTX:  %[[ncv:.*]] = alloca %struct.NCV
  // NVPTX:  %[[vd:.*]] = alloca %struct.VD
  // NVPTX:  %[[ncf:.*]] = alloca %struct.NCF
  // NVPTX:  %[[ncfs:.*]] = alloca %struct.NCFS
  // NVPTX:  %[[utc:.*]] = alloca %struct.UTC
  // NVPTX:  %[[netc:.*]] = alloca %struct.NETC
  // NVPTX:  %[[ec_i_ec:.*]] = alloca %struct.EC_I_EC
  // NVPTX:  %[[ec_i_ec1:.*]] = alloca %struct.EC_I_EC1
  // NVPTX:  %[[t_v_t:.*]] = alloca %struct.T_V_T
  // NVPTX:  %[[t_b_nec:.*]] = alloca %struct.T_B_NEC
  // NVPTX:  %[[t_f_nec:.*]] = alloca %struct.T_F_NEC
  // NVPTX:  %[[t_fa_nec:.*]] = alloca %struct.T_FA_NEC
  // NVPTX:  %[[t_b_ned:.*]] = alloca %struct.T_B_NED
  // NVPTX:  %[[t_f_ned:.*]] = alloca %struct.T_F_NED
  // NVPTX:  %[[t_fa_ned:.*]] = alloca %struct.T_FA_NED
  // AMDGCN:  %[[ec:.*]] = addrspacecast %struct.EC addrspace(5)* %ec to %struct.EC*
  // AMDGCN:  %[[ed:.*]] = addrspacecast %struct.ED addrspace(5)* %ed to %struct.ED*
  // AMDGCN:  %[[ecd:.*]] = addrspacecast %struct.ECD addrspace(5)* %ecd to %struct.ECD*
  // AMDGCN:  %[[etc:.*]] = addrspacecast %struct.ETC addrspace(5)* %etc to %struct.ETC*
  // AMDGCN:  %[[uc:.*]] = addrspacecast %struct.UC addrspace(5)* %uc to %struct.UC*
  // AMDGCN:  %[[ud:.*]] = addrspacecast %struct.UD addrspace(5)* %ud to %struct.UD*
  // AMDGCN:  %[[eci:.*]] = addrspacecast %struct.ECI addrspace(5)* %eci to %struct.ECI*
  // AMDGCN:  %[[nec:.*]] = addrspacecast %struct.NEC addrspace(5)* %nec to %struct.NEC*
  // AMDGCN:  %[[ned:.*]] = addrspacecast %struct.NED addrspace(5)* %ned to %struct.NED*
  // AMDGCN:  %[[ncv:.*]] = addrspacecast %struct.NCV addrspace(5)* %ncv to %struct.NCV*
  // AMDGCN:  %[[vd:.*]] = addrspacecast %struct.VD addrspace(5)* %vd to %struct.VD*
  // AMDGCN:  %[[ncf:.*]] = addrspacecast %struct.NCF addrspace(5)* %ncf to %struct.NCF*
  // AMDGCN:  %[[ncfs:.*]] = addrspacecast %struct.NCFS addrspace(5)* %ncfs to %struct.NCFS*
  // AMDGCN:  %[[utc:.*]] = addrspacecast %struct.UTC addrspace(5)* %utc to %struct.UTC*
  // AMDGCN:  %[[netc:.*]] = addrspacecast %struct.NETC addrspace(5)* %netc to %struct.NETC*
  // AMDGCN:  %[[ec_i_ec:.*]] = addrspacecast %struct.EC_I_EC addrspace(5)* %ec_i_ec to %struct.EC_I_EC*
  // AMDGCN:  %[[ec_i_ec1:.*]] = addrspacecast %struct.EC_I_EC1 addrspace(5)* %ec_i_ec1 to %struct.EC_I_EC1*
  // AMDGCN:  %[[t_v_t:.*]] = addrspacecast %struct.T_V_T addrspace(5)* %t_v_t to %struct.T_V_T*
  // AMDGCN:  %[[t_b_nec:.*]] = addrspacecast %struct.T_B_NEC addrspace(5)* %t_b_nec to %struct.T_B_NEC*
  // AMDGCN:  %[[t_f_nec:.*]] = addrspacecast %struct.T_F_NEC addrspace(5)* %t_f_nec to %struct.T_F_NEC*
  // AMDGCN:  %[[t_fa_nec:.*]] = addrspacecast %struct.T_FA_NEC addrspace(5)* %t_fa_nec to %struct.T_FA_NEC*
  // AMDGCN:  %[[t_b_ned:.*]] = addrspacecast %struct.T_B_NED addrspace(5)* %t_b_ned to %struct.T_B_NED*
  // AMDGCN:  %[[t_f_ned:.*]] = addrspacecast %struct.T_F_NED addrspace(5)* %t_f_ned to %struct.T_F_NED*
  // AMDGCN:  %[[t_fa_ned:.*]] = addrspacecast %struct.T_FA_NED addrspace(5)* %t_fa_ned to %struct.T_FA_NED*

  T t;
  // CHECK-NOT: call
  EC ec;
  // CHECK:  call void @_ZN2ECC1Ev(%struct.EC* %[[ec]])
  ED ed;
  // CHECK-NOT: call
  ECD ecd;
  // CHECK:  call void @_ZN3ECDC1Ev(%struct.ECD* %[[ecd]])
  ETC etc;
  // CHECK:  call void @_ZN3ETCC1IJEEEDpT_(%struct.ETC* %[[etc]])
  UC uc;
  // undefined constructor -- not allowed
  // CHECK:  call void @_ZN2UCC1Ev(%struct.UC* %[[uc]])
  UD ud;
  // undefined destructor -- not allowed
  // CHECK-NOT: call
  ECI eci;
  // empty constructor w/ initializer list -- not allowed
  // CHECK:  call void @_ZN3ECIC1Ev(%struct.ECI* %[[eci]])
  NEC nec;
  // non-empty constructor -- not allowed
  // CHECK:  call void @_ZN3NECC1Ev(%struct.NEC* %[[nec]])
  // non-empty destructor -- not allowed
  NED ned;
  // no-constructor,  virtual method -- not allowed
  // CHECK:  call void @_ZN3NCVC1Ev(%struct.NCV* %[[ncv]])
  NCV ncv;
  // CHECK-NOT: call
  VD vd;
  // CHECK:  call void @_ZN2VDC1Ev(%struct.VD* %[[vd]])
  NCF ncf;
  // CHECK:   call void @_ZN3NCFC1Ev(%struct.NCF* %[[ncf]])
  NCFS ncfs;
  // CHECK:  call void @_ZN4NCFSC1Ev(%struct.NCFS* %[[ncfs]])
  UTC utc;
  // CHECK:  call void @_ZN3UTCC1IJEEEDpT_(%struct.UTC* %[[utc]])
  NETC netc;
  // CHECK:  call void @_ZN4NETCC1IJEEEDpT_(%struct.NETC* %[[netc]])
  T_B_T t_b_t;
  // CHECK-NOT: call
  T_F_T t_f_t;
  // CHECK-NOT: call
  T_FA_T t_fa_t;
  // CHECK-NOT: call
  EC_I_EC ec_i_ec;
  // CHECK:  call void @_ZN7EC_I_ECC1Ev(%struct.EC_I_EC* %[[ec_i_ec]])
  EC_I_EC1 ec_i_ec1;
  // CHECK:  call void @_ZN8EC_I_EC1C1Ev(%struct.EC_I_EC1* %[[ec_i_ec1]])
  T_V_T t_v_t;
  // CHECK:  call void @_ZN5T_V_TC1Ev(%struct.T_V_T* %[[t_v_t]])
  T_B_NEC t_b_nec;
  // CHECK:  call void @_ZN7T_B_NECC1Ev(%struct.T_B_NEC* %[[t_b_nec]])
  T_F_NEC t_f_nec;
  // CHECK:  call void @_ZN7T_F_NECC1Ev(%struct.T_F_NEC* %[[t_f_nec]])
  T_FA_NEC t_fa_nec;
  // CHECK:  call void @_ZN8T_FA_NECC1Ev(%struct.T_FA_NEC* %[[t_fa_nec]])
  T_B_NED t_b_ned;
  // CHECK-NOT: call
  T_F_NED t_f_ned;
  // CHECK-NOT: call
  T_FA_NED t_fa_ned;
  // CHECK-NOT: call
  static __shared__ EC s_ec;
  // CHECK-NOT: call void @_ZN2ECC1Ev(%struct.EC* addrspacecast (%struct.EC addrspace(3)* @_ZZ2dfvE4s_ec to %struct.EC*))
  static __shared__ ETC s_etc;
  // CHECK-NOT: call void @_ZN3ETCC1IJEEEDpT_(%struct.ETC* addrspacecast (%struct.ETC addrspace(3)* @_ZZ2dfvE5s_etc to %struct.ETC*))

  static const int const_array[] = {1, 2, 3, 4, 5};
  static const int const_int = 123;

  // anchor point separating constructors and destructors
  df(); // CHECK: call void @_Z2dfv()

  // Verify that we only call non-empty destructors
  // CHECK-NEXT: call void @_ZN8T_FA_NEDD1Ev(%struct.T_FA_NED* %[[t_fa_ned]])
  // CHECK-NEXT: call void @_ZN7T_F_NEDD1Ev(%struct.T_F_NED* %[[t_f_ned]])
  // CHECK-NEXT: call void @_ZN7T_B_NEDD1Ev(%struct.T_B_NED* %[[t_b_ned]])
  // CHECK-NEXT: call void @_ZN2VDD1Ev(%struct.VD* %[[vd]])
  // CHECK-NEXT: call void @_ZN3NEDD1Ev(%struct.NED* %[[ned]])
  // CHECK-NEXT: call void @_ZN2UDD1Ev(%struct.UD* %[[ud]])
  // CHECK-NEXT: call void @_ZN3ECDD1Ev(%struct.ECD* %[[ecd]])
  // CHECK-NEXT: call void @_ZN2EDD1Ev(%struct.ED* %[[ed]])

  // CHECK-NEXT: ret void
}

// We should not emit global init function.
// CHECK-NOT: @_GLOBAL__sub_I
