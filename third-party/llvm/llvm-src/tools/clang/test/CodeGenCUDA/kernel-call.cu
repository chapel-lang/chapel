#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -emit-llvm %s -o - | FileCheck %s --check-prefixes=CUDA,CHECK
// RUN: %clang_cc1 -x hip -emit-llvm %s -o - | FileCheck %s --check-prefixes=HIP,CHECK


#include "Inputs/hip/hip_runtime.h"

// CHECK-LABEL: define{{.*}}g1
// HIP: call{{.*}}hipSetupArgument
// HIP: call{{.*}}hipLaunchByPtr
// CUDA: call{{.*}}hipSetupArgument
// CUDA: call{{.*}}hipLaunchByPtr
__global__ void g1(int x) {}

// CHECK-LABEL: define{{.*}}main
int main(void) {
  // HIP: call{{.*}}hipConfigureCall
  // CUDA: call{{.*}}hipConfigureCall
  // CHECK: icmp
  // CHECK: br
  // CHECK: call{{.*}}g1
  g1<<<1, 1>>>(42);
}
