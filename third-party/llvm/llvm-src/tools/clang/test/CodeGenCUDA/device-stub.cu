#include "hip/hip_runtime.h"
// RUN: echo "GPU binary would be here" > %t
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -fcuda-include-gpubinary %t -o - \
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s --check-prefixes=ALL,NORDC,CUDA,CUDANORDC
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -fcuda-include-gpubinary %t -o -  -DNOGLOBALS \
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s -check-prefixes=NOGLOBALS,CUDANOGLOBALS
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -fcuda-rdc -fcuda-include-gpubinary %t -o - \
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s --check-prefixes=ALL,RDC,CUDA,CUDARDC
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s -o - \
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s -check-prefix=NOGPUBIN

// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -fcuda-include-gpubinary %t -o - -x hip\
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s --check-prefixes=ALL,NORDC,HIP
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -fcuda-include-gpubinary %t -o -  -DNOGLOBALS -x hip \
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s -check-prefixes=NOGLOBALS,HIPNOGLOBALS
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -fcuda-rdc -fcuda-include-gpubinary %t -o - -x hip \
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s --check-prefixes=ALL,NORDC,HIP
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s -o - -x hip\
// RUN:   | FileCheck -allow-deprecated-dag-overlap %s -check-prefix=NOGPUBIN

#include "Inputs/hip/hip_runtime.h"

#ifndef NOGLOBALS
// ALL-DAG: @device_var = internal global i32
__device__ int device_var;

// ALL-DAG: @constant_var = internal global i32
__constant__ int constant_var;

// ALL-DAG: @shared_var = internal global i32
__shared__ int shared_var;

// Make sure host globals don't get internalized...
// ALL-DAG: @host_var = global i32
int host_var;
// ... and that extern vars remain external.
// ALL-DAG: @ext_host_var = external global i32
extern int ext_host_var;

// Shadows for external device-side variables are *definitions* of
// those variables.
// ALL-DAG: @ext_device_var = internal global i32
extern __device__ int ext_device_var;
// ALL-DAG: @ext_device_var = internal global i32
extern __constant__ int ext_constant_var;

void use_pointers() {
  int *p;
  p = &device_var;
  p = &constant_var;
  p = &shared_var;
  p = &host_var;
  p = &ext_device_var;
  p = &ext_constant_var;
  p = &ext_host_var;
}

// Make sure that all parts of GPU code init/cleanup are there:
// * constant unnamed string with the kernel name
// ALL: private unnamed_addr constant{{.*}}kernelfunc{{.*}}\00"
// * constant unnamed string with GPU binary
// HIP: @[[FATBIN:__hip_fatbin]] = external constant i8, section ".hip_fatbin"
// CUDA: @[[FATBIN:.*]] = private constant{{.*GPU binary would be here.*}}\00",
// CUDANORDC-SAME: section ".nv_fatbin", align 8
// CUDARDC-SAME: section "__nv_relfatbin", align 8
// * constant struct that wraps GPU binary
// ALL: @__[[PREFIX:cuda|hip]]_fatbin_wrapper = internal constant
// ALL-SAME: { i32, i32, i8*, i8* }
// CUDA-SAME: { i32 1180844977, i32 1,
// HIP-SAME: { i32 1212764230, i32 1,
// CUDA-SAME: i8* getelementptr inbounds ({{.*}}@[[FATBIN]], i64 0, i64 0),
// HIP-SAME:  i8* @[[FATBIN]],
// ALL-SAME: i8* null }
// CUDA-SAME: section ".nvFatBinSegment"
// HIP-SAME: section ".hipFatBinSegment"
// * variable to save GPU binary handle after initialization
// CUDANORDC: @__[[PREFIX]]_gpubin_handle = internal global i8** null
// HIP: @__[[PREFIX]]_gpubin_handle = linkonce global i8** null
// * constant unnamed string with NVModuleID
// RDC: [[MODULE_ID_GLOBAL:@.*]] = private constant
// CUDARDC-SAME: c"[[MODULE_ID:.+]]\00", section "__nv_module_id", align 32
// * Make sure our constructor was added to global ctor list.
// ALL: @llvm.global_ctors = appending global {{.*}}@__[[PREFIX]]_module_ctor
// * Alias to global symbol containing the NVModuleID.
// RDC: @__fatbinwrap[[MODULE_ID]] = alias { i32, i32, i8*, i8* }
// RDC-SAME: { i32, i32, i8*, i8* }* @__[[PREFIX]]_fatbin_wrapper

// Test that we build the correct number of calls to hipSetupArgument followed
// by a call to hipLaunchByPtr.

// ALL: define{{.*}}kernelfunc
// ALL: call{{.*}}[[PREFIX]]SetupArgument
// ALL: call{{.*}}[[PREFIX]]SetupArgument
// ALL: call{{.*}}[[PREFIX]]SetupArgument
// ALL: call{{.*}}[[PREFIX]]Launch
__global__ void kernelfunc(int i, int j, int k) {}

// Test that we've built correct kernel launch sequence.
// ALL: define{{.*}}hostfunc
// ALL: call{{.*}}[[PREFIX]]ConfigureCall
// ALL: call{{.*}}kernelfunc
void hostfunc(void) { kernelfunc<<<1, 1>>>(1, 1, 1); }
#endif

// Test that we've built a function to register kernels and global vars.
// ALL: define internal void @__[[PREFIX]]_register_globals
// ALL: call{{.*}}[[PREFIX]]RegisterFunction(i8** %0, {{.*}}kernelfunc
// ALL-DAG: call{{.*}}[[PREFIX]]RegisterVar(i8** %0, {{.*}}device_var{{.*}}i32 0, i32 4, i32 0, i32 0
// ALL-DAG: call{{.*}}[[PREFIX]]RegisterVar(i8** %0, {{.*}}constant_var{{.*}}i32 0, i32 4, i32 1, i32 0
// ALL-DAG: call{{.*}}[[PREFIX]]RegisterVar(i8** %0, {{.*}}ext_device_var{{.*}}i32 1, i32 4, i32 0, i32 0
// ALL-DAG: call{{.*}}[[PREFIX]]RegisterVar(i8** %0, {{.*}}ext_constant_var{{.*}}i32 1, i32 4, i32 1, i32 0
// ALL: ret void

// Test that we've built a constructor.
// ALL: define internal void @__[[PREFIX]]_module_ctor

// In separate mode it calls __[[PREFIX]]RegisterFatBinary(&__[[PREFIX]]_fatbin_wrapper)
// HIP only register fat binary once.
// HIP: load i8**, i8*** @__hip_gpubin_handle
// HIP-NEXT: icmp eq i8** {{.*}}, null
// HIP-NEXT: br i1 {{.*}}, label %if, label %exit
// HIP: if:
// NORDC: call{{.*}}[[PREFIX]]RegisterFatBinary{{.*}}__[[PREFIX]]_fatbin_wrapper
//   .. stores return value in __[[PREFIX]]_gpubin_handle
// NORDC-NEXT: store{{.*}}__[[PREFIX]]_gpubin_handle
//   .. and then calls __[[PREFIX]]_register_globals
// HIP-NEXT: br label %exit
// HIP: exit:
// HIP-NEXT: load i8**, i8*** @__hip_gpubin_handle
// NORDC-NEXT: call void @__[[PREFIX]]_register_globals
// * In separate mode we also register a destructor.
// NORDC-NEXT: call i32 @atexit(void (i8*)* @__[[PREFIX]]_module_dtor)

// With relocatable device code we call __[[PREFIX]]RegisterLinkedBinary%NVModuleID%
// RDC: call{{.*}}__[[PREFIX]]RegisterLinkedBinary[[MODULE_ID]](
// RDC-SAME: __[[PREFIX]]_register_globals, {{.*}}__[[PREFIX]]_fatbin_wrapper
// RDC-SAME: [[MODULE_ID_GLOBAL]]

// Test that we've created destructor.
// NORDC: define internal void @__[[PREFIX]]_module_dtor
// NORDC: load{{.*}}__[[PREFIX]]_gpubin_handle
// CUDANORDC-NEXT: call void @__[[PREFIX]]UnregisterFatBinary
// HIP-NEXT: icmp ne i8** {{.*}}, null
// HIP-NEXT: br i1 {{.*}}, label %if, label %exit
// HIP: if:
// HIP-NEXT: call void @__[[PREFIX]]UnregisterFatBinary
// HIP-NEXT: store i8** null, i8*** @__hip_gpubin_handle
// HIP-NEXT: br label %exit
// HIP: exit:

// There should be no __[[PREFIX]]_register_globals if we have no
// device-side globals, but we still need to register GPU binary.
// Skip GPU binary string first.
// CUDANOGLOBALS: @{{.*}} = private constant{{.*}}
// HIPNOGLOBALS: @{{.*}} = external constant{{.*}}
// NOGLOBALS-NOT: define internal void @__{{.*}}_register_globals
// NOGLOBALS: define internal void @__[[PREFIX:cuda|hip]]_module_ctor
// NOGLOBALS: call{{.*}}[[PREFIX]]RegisterFatBinary{{.*}}__[[PREFIX]]_fatbin_wrapper
// NOGLOBALS-NOT: call void @__[[PREFIX]]_register_globals
// NOGLOBALS: define internal void @__[[PREFIX]]_module_dtor
// NOGLOBALS: call void @__[[PREFIX]]UnregisterFatBinary

// There should be no constructors/destructors if we have no GPU binary.
// NOGPUBIN-NOT: define internal void @__[[PREFIX]]_register_globals
// NOGPUBIN-NOT: define internal void @__[[PREFIX]]_module_ctor
// NOGPUBIN-NOT: define internal void @__[[PREFIX]]_module_dtor
