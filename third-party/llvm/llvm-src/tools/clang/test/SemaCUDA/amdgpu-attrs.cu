#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fsyntax-only -verify %s
#include "Inputs/hip/hip_runtime.h"


__attribute__((amdgpu_flat_work_group_size(32, 64)))
__global__ void flat_work_group_size_32_64() {}

__attribute__((amdgpu_waves_per_eu(2)))
__global__ void waves_per_eu_2() {}

__attribute__((amdgpu_waves_per_eu(2, 4)))
__global__ void waves_per_eu_2_4() {}

__attribute__((amdgpu_num_sgpr(32)))
__global__ void num_sgpr_32() {}

__attribute__((amdgpu_num_vgpr(64)))
__global__ void num_vgpr_64() {}


__attribute__((amdgpu_flat_work_group_size(32, 64), amdgpu_waves_per_eu(2)))
__global__ void flat_work_group_size_32_64_waves_per_eu_2() {}

__attribute__((amdgpu_flat_work_group_size(32, 64), amdgpu_waves_per_eu(2, 4)))
__global__ void flat_work_group_size_32_64_waves_per_eu_2_4() {}

__attribute__((amdgpu_flat_work_group_size(32, 64), amdgpu_num_sgpr(32)))
__global__ void flat_work_group_size_32_64_num_sgpr_32() {}

__attribute__((amdgpu_flat_work_group_size(32, 64), amdgpu_num_vgpr(64)))
__global__ void flat_work_group_size_32_64_num_vgpr_64() {}

__attribute__((amdgpu_waves_per_eu(2), amdgpu_num_sgpr(32)))
__global__ void waves_per_eu_2_num_sgpr_32() {}

__attribute__((amdgpu_waves_per_eu(2), amdgpu_num_vgpr(64)))
__global__ void waves_per_eu_2_num_vgpr_64() {}

__attribute__((amdgpu_waves_per_eu(2, 4), amdgpu_num_sgpr(32)))
__global__ void waves_per_eu_2_4_num_sgpr_32() {}

__attribute__((amdgpu_waves_per_eu(2, 4), amdgpu_num_vgpr(64)))
__global__ void waves_per_eu_2_4_num_vgpr_64() {}

__attribute__((amdgpu_num_sgpr(32), amdgpu_num_vgpr(64)))
__global__ void num_sgpr_32_num_vgpr_64() {}

__attribute__((amdgpu_flat_work_group_size(32, 64), amdgpu_waves_per_eu(2), amdgpu_num_sgpr(32)))
__global__ void flat_work_group_size_32_64_waves_per_eu_2_num_sgpr_32() {}

__attribute__((amdgpu_flat_work_group_size(32, 64), amdgpu_waves_per_eu(2), amdgpu_num_vgpr(64)))
__global__ void flat_work_group_size_32_64_waves_per_eu_2_num_vgpr_64() {}

__attribute__((amdgpu_flat_work_group_size(32, 64), amdgpu_waves_per_eu(2, 4), amdgpu_num_sgpr(32)))
__global__ void flat_work_group_size_32_64_waves_per_eu_2_4_num_sgpr_32() {}

__attribute__((amdgpu_flat_work_group_size(32, 64), amdgpu_waves_per_eu(2, 4), amdgpu_num_vgpr(64)))
__global__ void flat_work_group_size_32_64_waves_per_eu_2_4_num_vgpr_64() {}

__attribute__((amdgpu_flat_work_group_size(32, 64), amdgpu_waves_per_eu(2), amdgpu_num_sgpr(32), amdgpu_num_vgpr(64)))
__global__ void flat_work_group_size_32_64_waves_per_eu_2_num_sgpr_32_num_vgpr_64() {}

__attribute__((amdgpu_flat_work_group_size(32, 64), amdgpu_waves_per_eu(2, 4), amdgpu_num_sgpr(32), amdgpu_num_vgpr(64)))
__global__ void flat_work_group_size_32_64_waves_per_eu_2_4_num_sgpr_32_num_vgpr_64() {}

// expected-error@+2{{attribute 'reqd_work_group_size' can only be applied to an OpenCL kernel function}}
__attribute__((reqd_work_group_size(32, 64, 64)))
__global__ void reqd_work_group_size_32_64_64() {}

// expected-error@+2{{attribute 'work_group_size_hint' can only be applied to an OpenCL kernel function}}
__attribute__((work_group_size_hint(2, 2, 2)))
__global__ void work_group_size_hint_2_2_2() {}

// expected-error@+2{{attribute 'vec_type_hint' can only be applied to an OpenCL kernel function}}
__attribute__((vec_type_hint(int)))
__global__ void vec_type_hint_int() {}

// expected-error@+2{{attribute 'intel_reqd_sub_group_size' can only be applied to an OpenCL kernel function}}
__attribute__((intel_reqd_sub_group_size(64)))
__global__ void intel_reqd_sub_group_size_64() {}
