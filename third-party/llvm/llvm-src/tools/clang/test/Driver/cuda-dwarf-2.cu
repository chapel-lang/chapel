
#include <hip/hip_runtime.h>
// REQUIRES: clang-driver
//
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g -O0 --no-cuda-noopt-device-debug 2>&1 | \
// RUN:   FileCheck %s -check-prefix NO_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g -O3 2>&1 | \
// RUN:   FileCheck %s -check-prefix NO_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g -O3 --no-cuda-noopt-device-debug 2>&1 | \
// RUN:   FileCheck %s -check-prefix NO_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g0 2>&1 | \
// RUN:   FileCheck %s -check-prefix NO_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -ggdb0 -O3 --cuda-noopt-device-debug 2>&1 | \
// RUN:   FileCheck %s -check-prefix NO_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -ggdb1 2>&1 | \
// RUN:   FileCheck %s -check-prefix NO_DEBUG -check-prefix LINE_TABLE
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -gline-tables-only -O2 --cuda-noopt-device-debug 2>&1 | \
// RUN:   FileCheck %s -check-prefix NO_DEBUG -check-prefix LINE_TABLE

// NO_DEBUG-NOT: warning: debug
// LINE_TABLE-NOT: warning: debug
// NO_DEBUG: ptxas
// NO_DEBUG-NOT: "-g"
// LINE_TABLE: "-lineinfo"
// NO_DEBUG: fatbinary
// NO_DEBUG-NOT: "-g"

// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g 2>&1 | \
// RUN:   FileCheck %s -check-prefix HAS_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g -O0 --cuda-noopt-device-debug 2>&1 | \
// RUN:   FileCheck %s -check-prefix HAS_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g -O3 --cuda-noopt-device-debug 2>&1 | \
// RUN:   FileCheck %s -check-prefix HAS_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g2 2>&1 | \
// RUN:   FileCheck %s -check-prefix HAS_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -ggdb2 -O0 2>&1 | \
// RUN:   FileCheck %s -check-prefix HAS_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -g3 -O2 --cuda-noopt-device-debug 2>&1 | \
// RUN:   FileCheck %s -check-prefix HAS_DEBUG
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s -ggdb3 -O3 --cuda-noopt-device-debug 2>&1 | \
// RUN:   FileCheck %s -check-prefix HAS_DEBUG

// HAS_DEBUG-NOT: warning: debug
// HAS_DEBUG: "-fcuda-is-device"
// HAS_DEBUG-SAME: "-dwarf-version=2"
// HAS_DEBUG: ptxas
// HAS_DEBUG-SAME: "-g"
// HAS_DEBUG-SAME: "--dont-merge-basicblocks"
// HAS_DEBUG-SAME: "--return-at-end"
// HAS_DEBUG: fatbinary
// HAS_DEBUG-SAME: "-g"

