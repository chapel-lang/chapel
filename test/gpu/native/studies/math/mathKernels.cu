#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <hipcub/hipcub.hpp>
#include <math.h>

#ifndef mk_PREFIX
#define mk_PREFIX
#endif

#ifndef mk_LABEL
#define mk_LABEL ""
#endif

#ifndef mk_MATH_FUNC
#define mk_MATH_FUNC tanhf
#endif

#ifndef mk_SIZE
#define mk_SIZE 100'000'000
#endif

#define mk_FUNC_NAME_inner2(a, b) a ## b
#define mk_FUNC_NAME_inner1(a, b) mk_FUNC_NAME_inner2(a, b)
#define mk_FUNC_NAME(a) mk_FUNC_NAME_inner1(mk_PREFIX, a)

int mk_FUNC_NAME(ceil_div)(int dividend, int divisor) {
    return (dividend + divisor - 1) / divisor;
}
__global__ void mk_FUNC_NAME(kernel)(float* arr, int N, int iterations) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    float s = 0;
    for (int j = 0; j < iterations; j++)
      s += mk_MATH_FUNC(i);
    arr[i] = s;
  }
}

__global__ void mk_FUNC_NAME(init_seed)(hiprandState *state, int seed) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed, idx, 0, &state[idx]);
}
__global__ void mk_FUNC_NAME(fillRand)(hiprandState *state, float *arr, int N) {
 int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    arr[i] = hiprand_uniform(&state[i]);
  }
}

void mk_FUNC_NAME(main)(int printTime, int correctness, int iterations) {
  int deviceIdx = 0;
  hipSetDevice(deviceIdx);
  int N = mk_SIZE;
  int block_size = 256;
  float* arr;
  hipMalloc(&arr, N * sizeof(float));
  float grid_size = mk_FUNC_NAME(ceil_div)(N, block_size);

  hiprandState *d_state;
  hipMalloc(&d_state, (N * sizeof(hiprandState)));
  mk_FUNC_NAME(init_seed)<<<grid_size, block_size>>>(d_state, time(NULL));
  mk_FUNC_NAME(fillRand)<<<grid_size, block_size>>>(d_state, arr, N);


  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, nullptr);
  mk_FUNC_NAME(kernel)<<<grid_size, block_size>>>(arr, N, iterations);
  hipDeviceSynchronize(); // Wait for the GPU to finish
  hipEventRecord(stop, nullptr);
  hipEventSynchronize(start);
  hipEventSynchronize(stop);

  float elapsed;
  hipEventElapsedTime(&elapsed, start, stop);
  if (printTime)
    printf(mk_LABEL" Time: %f ms\n", elapsed);

  if (correctness) {
    float* sum;
    hipMalloc(&sum, sizeof(float));
    void *temp_storage = nullptr;
    size_t n_temp_storage = 0;
    hipcub::DeviceReduce::Sum(temp_storage, n_temp_storage, arr, sum, N);
    hipMalloc(&temp_storage, n_temp_storage);
    hipcub::DeviceReduce::Sum(temp_storage, n_temp_storage, arr, sum, N);
    float sum_host;
    hipMemcpy(&sum_host, sum, sizeof(float), hipMemcpyDeviceToHost);
    printf(mk_LABEL" Sum: %f\n", sum_host);
  }
}
